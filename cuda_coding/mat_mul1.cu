#include <hip/hip_runtime.h>

#include <stdio.h>

hipError_t matmul(int* c, int* a, int* b, int size[4], dim3 threads, dim3 blocks);

__global__ void mulKernel(int* c, int* a, int* b, int M, int N, int cols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < M && col < N) {
		int tmp = 0;
		for (int i = 0; i < cols; i++) {
			tmp += a[N * row + i] * b[N * i + col];
		}
		c[N * row + col] = tmp;
	}
}

int main() {
	const int m = 2;
	const int n = 3;
	int a2d[m][n] = { {4, 2, 3}, {5, 1, 6} };
	int b2d[n][m] = { {6, 6}, {7, 2}, {3,9} };
	int a[m * n];
	int b[n * m];
	int i, j, k;
	k = 0;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			a[k] = a2d[i][j];
			printf("%d ", a[k]);
			k++;
		}
		printf("\n");
	}
	k = 0;
	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			b[k] = b2d[i][j];
			printf("%d ", b[k]);
			k++;
		}
		printf("\n");
	}
	int c[m * m] = { 0 };

	int threads = 2;
	int blocks = (m*m + threads - 1) / threads;
	dim3 THREADS(threads, threads);
	dim3 BLOCKS(blocks, blocks);
	int dim[4] = { m, n, n, m };
	hipError_t status = matmul(c, a, b, dim, THREADS, BLOCKS);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed\n");
		return 1;
	}

	k = 0;
	for (i = 0; i < m; i++) {
		for (j = 0; j < m; j++) {
			printf("%d ", c[k]);
			k++;
		}
		printf("\n");
	}

	status = hipDeviceReset();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to reset the device\n");
		return 1;
	}

	return 0;
}

hipError_t matmul(int* c, int* a, int* b, int size[4], dim3 threads, dim3 blocks) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t status;

	status = hipSetDevice(0);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to set a device\n");
		goto Error;
	}

	status = hipMalloc((void**)&dev_a, size[0]*size[1] * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory\n");
		goto Error;
	}
	status = hipMalloc((void**)&dev_b, size[2] * size[3] * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory\n");
		goto Error;
	}
	status = hipMalloc((void**)&dev_c, size[0] * size[3] * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory\n");
		goto Error;
	}

	status = hipMemcpy(dev_a, a, size[0] * size[1] * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from host to GPU buffer\n");
		goto Error;
	}
	status = hipMemcpy(dev_b, b, size[2] * size[3] * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from host to GPU buffer\n");
		goto Error;
	}

	mulKernel<<<blocks, threads>>>(dev_c, dev_a, dev_b, size[0], size[3], size[1]);

	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Last error message %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to synchronize deivce \n");
		goto Error;
	}

	status = hipMemcpy(c, dev_c, size[0] * size[3] * sizeof(int), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from GPU buffer to host\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return status;
}
#include <hip/hip_runtime.h>


#include <stdio.h>

hipError_t MatMul(int* c, int* a, int* b, unsigned int size, int m, int n, dim3 THREADS, dim3 BLOCKS);

__global__ void multKernel(int* c, int* a, int* b, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < N && col < N) {
		int tmp = 0;
		for (int i = 0; i < N; i++) {
			tmp += a[N * row + i] * b[N * i + col];
		}
		c[N * row + col] = tmp;
		printf("%d, %d = %d\n", row ,col, tmp);
	}
}

int main() {
	const int m = 2, n = 2;
	const int size = m * n;
	int i, j, k;
	int one2d[m][n] = { {6, 4}, {7, 7} };
	int two2d[m][n] = { {2, 9}, {4, 6} };
	int one[size];
	int two[size];
	k = 0;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			one[k] = one2d[i][j];
			two[k] = two2d[i][j];
			k++;
		}
	}
	int res[size] = { 0 };
	int threads = 1;
	int blocks = (size + threads - 1) / threads;
	dim3 THREADS(threads, threads);
	dim3 BLOCKS(blocks, blocks);
	hipError_t status = MatMul(res, one, two, size, m, n, THREADS, BLOCKS);

	if (status != hipSuccess) {
		fprintf(stderr, "Failed to multiple quadratic matrix\n");
		return 1;
	}
	k = 0;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			printf("%d ", res[k]);
			k++;
		}
		printf("\n");
	}

	status = hipDeviceReset();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to reset CUDA device\n");
		return 1;
	}

	return 0;
}


hipError_t MatMul(int* c, int* a, int* b, unsigned int size, int m, int n, dim3 THREADS, dim3 BLOCKS) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t status;

	status = hipSetDevice(0);
	if (status != hipSuccess) {
		fprintf(stderr, "CUDA device not found\n");
		goto Error;
	}

	status = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Memory allocation error\n");
		goto Error;
	}
	status = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Memory allocation error\n");
		goto Error;
	}
	status = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "Memory allocation error\n");
		goto Error;
	}

	status = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from host to GPU buffer\n");
		goto Error;
	}
	status = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from host to GPU buffer\n");
		goto Error;
	}

	multKernel<<<BLOCKS,THREADS>>>(dev_c, dev_a, dev_b, m);

	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Error: %s \n", hipGetErrorString(status));
		goto Error;
	}

	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to synchronize device, error no. :%d\n", status);
		goto Error;
	}

	status = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from GPU buffer to host array\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_b);
	return status;
}
